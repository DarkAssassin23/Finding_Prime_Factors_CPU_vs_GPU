#include "hip/hip_runtime.h"
#include <iostream>
#include <cstdint>
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <assert.h>
#include <math.h>
#include <thread>
#include <random>
#include <vector>
#include "primes.h"

bool isPrime(int number)
{
    if(number < 2) 
        return false;
    if(number == 2) 
        return true;
    if(number % 2 == 0) 
        return false;
    for (int x = 3; (x*x) < number; x+=2)
        if(number % x == 0) 
            return false;
    
    return true;
}

void genPrimes(int minRange, int maxRange, std::vector<int> *primeList)
{
    for(int x = minRange; x < maxRange; x++)
        if(isPrime(x))
            primeList->push_back(x);
}

std::vector<int> multithreadPrimeGen(int maxNumber)
{
    const auto processor_count = std::thread::hardware_concurrency();
    std::vector<std::thread> threads;
    std::vector<std::vector<int>> totalPrimesListSegment(processor_count);
    int range = maxNumber / processor_count;

    for(int x=0;x<processor_count;x++)
    {
        int start = x * range;
        int end = (x+1) * range;
        threads.push_back(std::thread(genPrimes, start, end, &totalPrimesListSegment.at(x)));
    }

    for(int x=0;x<processor_count;x++)
        threads.at(x).join();

    std::vector<int> primeList;
    for(auto x : totalPrimesListSegment)
        primeList.insert(primeList.end(), x.begin(), x.end());

    return primeList;
}

uint32_t genRandomNum(uint32_t lower, uint32_t upper)
{
    std::random_device dev;
    std::mt19937 rng(dev());
    std::uniform_int_distribution<std::mt19937::result_type> genRandom(lower, upper);
    
    return genRandom(rng);
}

uint64_t getRandomProduct(std::vector<uint32_t> *primes)
{
    uint32_t p1 = genRandomNum(0, primes->size());
    uint32_t p2 = p1;
    do
    {
        p2 = genRandomNum(0, primes->size());
    }while (p2 == p1);

    return (static_cast<uint64_t>(primes->at(p1)) * static_cast<uint64_t>(primes->at(p2)));
}

// Barring minor changes, the follow code below is courtesy of the
// following stackoverlow post: https://stackoverflow.com/q/15622196
#define MAX_BLOCKS 256
#define THREADS_PER_BLOCK 256 //Must be a power of 2
#define BLOCK_SPACE 2 * THREADS_PER_BLOCK
#define MIN_PRIMES 10000000 // Minimum primes to generate

__global__ void initialize(uint32_t* isPrime, uint32_t n) 
{
    uint32_t idx = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
    uint32_t step = gridDim.x * THREADS_PER_BLOCK;
    uint32_t i;
    for (i = idx; i <= 1; i += step)
        isPrime[i] = 0;

    for (; i < n; i += step)
        isPrime[i] = 1;
}

__global__ void clearMultiples(uint32_t* isPrime, uint32_t* primeList, 
                               uint32_t startInd, uint32_t endInd, 
                               uint32_t n) 
{
    uint32_t yidx = blockIdx.y * blockDim.y + threadIdx.y;
    uint32_t xidx = blockIdx.x * blockDim.x + threadIdx.x;
    uint32_t ystep = gridDim.y * blockDim.y;
    uint32_t xstep = gridDim.x * blockDim.x;
    for (uint32_t pnum = startInd + yidx; pnum < endInd; pnum += ystep) 
    {
        uint32_t p = primeList[pnum];
        uint32_t pstart = p * (p + xidx);
        uint32_t pstep = p * xstep;
        for (uint32_t i = pstart; i < n; i += pstep)
            isPrime[i] = 0;

    }
}

__device__ void makeCounts(uint32_t* isPrime, uint32_t* addend, 
                           uint32_t start, uint32_t stop) 
{
    __shared__ uint32_t tmpCounts[BLOCK_SPACE];
    __shared__ uint32_t dumbCounts[BLOCK_SPACE];
    uint32_t idx = threadIdx.x;
    tmpCounts[idx] = ((start + idx) < stop) ? isPrime[start + idx] : 0;
    __syncthreads();
    uint32_t numEntries = THREADS_PER_BLOCK;
    uint32_t cstart = 0;
    while (numEntries > 1) 
    {
        uint32_t prevStart = cstart;
        cstart += numEntries;
        numEntries /= 2;
        if (idx < numEntries)
        {
            uint32_t i1 = idx * 2 + prevStart;
            tmpCounts[idx + cstart] = tmpCounts[i1] + tmpCounts[i1 + 1];
        }
        __syncthreads();
    }
    if (idx == 0)
    {
        dumbCounts[cstart] = tmpCounts[cstart];
        tmpCounts[cstart] = 0;
    }
    while (cstart > 0) 
    {
        uint32_t prevStart = cstart;
        cstart -= numEntries * 2;
        if (idx < numEntries) 
        {
            uint32_t v1 = tmpCounts[idx + prevStart];
            uint32_t i1 = idx * 2 + cstart;
            tmpCounts[i1 + 1] = tmpCounts[i1] + v1;
            tmpCounts[i1] = v1;
            dumbCounts[i1] = dumbCounts[i1 + 1] = dumbCounts[idx + prevStart];
        }
        numEntries *= 2;
        __syncthreads();
    }
    if (start + idx < stop) 
    {
        isPrime[start + idx] = tmpCounts[idx];
        addend[start + idx] = dumbCounts[idx];
    }
}

__global__ void createCounts(uint32_t* isPrime, uint32_t* addend, 
                             uint32_t lb, uint32_t ub) 
{
    uint32_t step = gridDim.x * THREADS_PER_BLOCK;
    for (uint32_t i = lb + blockIdx.x * THREADS_PER_BLOCK; i < ub; i += step) 
    {
        uint32_t start = i;
        uint32_t stop = min(i + step, ub);
        makeCounts(isPrime, addend, start, stop);
    }
}

__global__ void sumCounts(uint32_t* isPrime, uint32_t* addend, 
                          uint32_t lb, uint32_t ub, uint32_t* totalsum) 
{
    uint32_t idx = blockIdx.x;
    uint32_t s = 0;
    for (uint32_t i = lb + idx; i < ub; i += THREADS_PER_BLOCK) 
    {
        isPrime[i] += s;
        s += addend[i];
    }
    if (idx == 0)
        *totalsum = s;
}

__global__ void condensePrimes(uint32_t* isPrime, uint32_t* primeList, 
                               uint32_t lb, uint32_t ub,
                               uint32_t primeStartInd, uint32_t primeCount) 
{
    uint32_t idx = blockIdx.x * THREADS_PER_BLOCK + threadIdx.x;
    uint32_t step = gridDim.x * THREADS_PER_BLOCK;
    for (uint32_t i = lb + idx; i < ub; i += step)
    {
        uint32_t term = isPrime[i];
        uint32_t nextTerm = i + 1 == ub ? primeCount : isPrime[i + 1];
        if (term < nextTerm)
            primeList[primeStartInd + term] = i;
    }
}

std::vector<uint32_t> genPrimesGPU(void)
{
    // Get device properties of GPU 0 to get the amount of memory available
    // and calculate how many primes we can generate
    hipDeviceProp_t prop;
    hipGetDeviceProperties(&prop, 0);

    // Half the memory for `isPrime` and the other half for `addend`
    // plus buffer to not allocate more VRAM than the card has
    const uint32_t MAX_RANGE = (prop.totalGlobalMem * 0.45f) 
                                / sizeof(uint32_t);
    const uint32_t n = max((uint32_t)MIN_PRIMES, MAX_RANGE);
    std::cout<<"Generating prime numbers from 0 - "<<n<<"\n";
    
    // If the GPU doesn't have enough memory to generate the minimum
    // number of primes, fall back to generating them with the CPU
    if(MAX_RANGE < MIN_PRIMES)
    {
        std::vector<int> tmp = multithreadPrimeGen((int) n);
        std::vector<uint32_t> result;
        std::copy(tmp.begin(), tmp.end(), 
                  back_inserter(result));

        return result;
    }

    uint32_t *isPrime, *addend, *numPrimes, *primeList;
    hipError_t t = hipMalloc((void**) &isPrime, n * sizeof(uint32_t));
    assert(t == hipSuccess);
    
    t = hipMalloc(&addend, n * sizeof(uint32_t));
    assert(t == hipSuccess);
    
    t = hipMalloc(&numPrimes, sizeof(uint32_t));
    assert(t == hipSuccess);
    
    uint32_t primeBound = 2 * n / log(n);
    t = hipMalloc(&primeList, primeBound * sizeof(uint32_t));
    assert(t == hipSuccess);
    
    uint32_t numBlocks = min((uint32_t)MAX_BLOCKS,
                             (n + THREADS_PER_BLOCK - 1) 
                             / THREADS_PER_BLOCK);
    initialize<<<numBlocks, THREADS_PER_BLOCK>>>(isPrime, n);
    t = hipDeviceSynchronize();
    assert(t == hipSuccess);

    uint32_t bound = (uint32_t) ceil(sqrt(n));
    uint32_t lb;
    uint32_t ub = 2;
    uint32_t primeStartInd = 0;
    uint32_t primeEndInd = 0;

    while (ub < n) 
    {
        if (primeEndInd > primeStartInd) 
        {
            uint32_t lowprime;
            t = hipMemcpy(&lowprime, primeList + primeStartInd, 
                           sizeof(uint32_t), hipMemcpyDeviceToHost);
            assert(t == hipSuccess);

            uint32_t numcols = n / lowprime;
            uint32_t numrows = primeEndInd - primeStartInd;
            uint32_t threadx = min(numcols, (uint32_t)THREADS_PER_BLOCK);
            uint32_t thready = min(numrows, THREADS_PER_BLOCK / threadx);
            uint32_t blockx = min(numcols / threadx, (uint32_t) MAX_BLOCKS);
            uint32_t blocky = min(numrows / thready, MAX_BLOCKS / blockx);

            dim3 gridsize(blockx, blocky);
            dim3 blocksize(threadx, thready);
            clearMultiples<<<gridsize, blocksize>>>(isPrime, primeList,
                                                    primeStartInd, 
                                                    primeEndInd, n);
            t = hipDeviceSynchronize();
            assert(t == hipSuccess);
        }
        lb = ub;
        ub *= 2;
        if (lb >= bound)
            ub = n;
        
        numBlocks = min((uint32_t)MAX_BLOCKS,
                        (ub - lb + THREADS_PER_BLOCK - 1) 
                        / THREADS_PER_BLOCK);

        createCounts<<<numBlocks, THREADS_PER_BLOCK>>>(isPrime, addend, 
                                                       lb, ub);
        t = hipDeviceSynchronize();
        assert(t == hipSuccess);

        sumCounts<<<THREADS_PER_BLOCK, 1>>>(isPrime, addend, lb, ub, 
                                            numPrimes);
        t = hipDeviceSynchronize();
        assert(t == hipSuccess);

        uint32_t primeCount;
        t = hipMemcpy(&primeCount, numPrimes, sizeof(uint32_t),
                       hipMemcpyDeviceToHost);
        assert(t == hipSuccess);
        assert(primeCount > 0);

        primeStartInd = primeEndInd;
        primeEndInd += primeCount;
        condensePrimes<<<numBlocks, THREADS_PER_BLOCK>>>(isPrime, primeList,
                                                         lb, ub, 
                                                         primeStartInd, 
                                                         primeCount);
        t = hipDeviceSynchronize();
        assert(t == hipSuccess);
    }

    uint32_t *finalprimes = (uint32_t *) malloc(primeEndInd 
                                                * sizeof(uint32_t));
    t = hipMemcpy(finalprimes, primeList, primeEndInd * sizeof(uint32_t),
                   hipMemcpyDeviceToHost);
    assert(t == hipSuccess);

    t = hipFree(isPrime);
    assert(t == hipSuccess);

    t = hipFree(addend);
    assert(t == hipSuccess);

    t = hipFree(numPrimes);
    assert(t == hipSuccess);

    t = hipFree(primeList);
    assert(t == hipSuccess);
    
    std::vector<uint32_t> result;
    std::copy(&finalprimes[0], &finalprimes[primeEndInd], 
              back_inserter(result));

    free(finalprimes);

    return result;
}
